#include "hip/hip_runtime.h"
//22.4.8
/*
	C:\"Program Files\NVIDIA GPU Computing Toolkit"\CUDA\v10.2\bin\nvcc.exe .\Mandelbrot-Cuda.cu -o .\Mandelbrot-Cuda.exe -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.24.28314\bin\Hostx64\x64"  -std=c++14 -w -I"I:\ProgramProjects\DEV64-VC\include" -I"I:\ProgramProjects\PAL\PAL_Library"  -L"I:\ProgramProjects\DEV64-VC\lib" -lSDL2main -lSDL2 -lSDL2_ttf -lUser32 -Xcompiler /w34819
*/
#include "../FractalTemplate.cpp"
#include <complex>

#include <hip/hip_runtime.h>
#include <>

#define IterationColorCount() 256
#define IterationColor(t) (255-(t)|255-(t)<<8|255<<16|255<<24)

__global__ void GetFractalInKernel(Uint32 *data,const int W,const int H,const int Group,const double EscapeRadius)
{
	int y=threadIdx.x+Group*1024;
	int x=blockIdx.x;
	
	double Cx=x*2.0/W-1-0.5,
		   Cy=y*2.0/H-1,
		   Zx=0,
		   Zy=0;
	int t=0;
	while (t<IterationColorCount()&&Zx*Zx+Zy*Zy<=EscapeRadius*EscapeRadius)
	{
		double nx=Zx*Zx-Zy*Zy+Cx,
			   ny=2*Zx*Zy+Cy;
		Zx=nx;
		Zy=ny;
		++t;
	}
	if (t>=IterationColorCount())
		t=0;
	data[y*W+x]=IterationColor(t);
}

FractalParameter EscapeRadius("EscapeRaius",2,0.05,10);

void Fractal(const Posize &ps)
{
	Uint32 *Buffer=new Uint32[ps.w*ps.h];
	
	Uint32 *DeviceData=nullptr;
	hipMalloc((void**)&DeviceData,ps.Size()*sizeof(Uint32));
	for (int i=0;i<(ps.h+1023)/1024;++i)
	{
		dim3 grid(ps.w);
		dim3 threads(min(ps.h-i*1024,1024));
		GetFractalInKernel << <grid, threads >> >(DeviceData,ps.w,ps.h,i,EscapeRadius);
		hipDeviceSynchronize();
	}
	hipMemcpy(Buffer,DeviceData,ps.Size()*sizeof(Uint32),hipMemcpyDeviceToHost);
    hipFree(DeviceData);
	
	SDL_Surface *sur=SDL_CreateRGBSurfaceWithFormatFrom((void*)Buffer,ps.w,ps.h,32,ps.w*4,SDL_PIXELFORMAT_RGBA32);
	SDL_SetSurfaceBlendMode(sur,SDL_BLENDMODE_BLEND);
	CurrentWindow->RenderCopy(CreateTextureFromSurfaceAndDelete(sur),ps.ToOrigin(),ps);
	DELETEtoNULL(Buffer);
}

void FractalInit()
{
	SetFractalName(PUIT("Mandelbrot EscapeTimeAlgorithm"));
	EscapeRadius.InitWidgets();
}

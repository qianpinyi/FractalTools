#include "hip/hip_runtime.h"
//22.4.8
/*
	C:\"Program Files\NVIDIA GPU Computing Toolkit"\CUDA\v10.2\bin\nvcc.exe .\Mandelbrot-Cuda.cu -o .\Mandelbrot-Cuda.exe -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\Community\VC\Tools\MSVC\14.24.28314\bin\Hostx64\x64"  -std=c++14 -w -I"I:\ProgramProjects\DEV64-VC\include" -I"I:\ProgramProjects\PAL\PAL_Library"  -L"I:\ProgramProjects\DEV64-VC\lib" -lSDL2main -lSDL2 -lSDL2_ttf -lUser32 -Xcompiler /w34819
*/
#include "../FractalTemplate.cpp"

#include <hip/hip_runtime.h>
#include <>

#define IterationColor(t) \
	~(((t)>>0)&0xff|((t)>>1&0xff)<<8|((t)>>2&0xff)<<16|0)
//	((255-(t)%256)%256|(255-(t)*2%256)%256<<8|(255)%256<<16|255<<24)

__global__ void GetFractalInKernel(Uint32 *data,const int W,const int H,const int Group,const double EscapeRadius,const double Scale,const double ShiftX,const double ShiftY,const int IterationColorCount)
{
	int y=threadIdx.x+Group*1024;
	int x=blockIdx.x;
	
	double Cx=(x*2.0/W-1)/Scale+ShiftX,
		   Cy=(y*2.0/H-1)/Scale+ShiftY,
		   Zx=0,
		   Zy=0;
	int t=0;
	while (t<IterationColorCount&&Zx*Zx+Zy*Zy<=EscapeRadius*EscapeRadius)
	{
		double nx=Zx*Zx-Zy*Zy+Cx,
			   ny=2*Zx*Zy+Cy;
		Zx=nx;
		Zy=ny;
		++t;
	}
	if (t>=IterationColorCount)
		t=0;
	data[y*W+x]=IterationColor(t);
}

FractalParameter EscapeRadius("EscapeRaius",2,0.05,10),
				 Scale("Scale",1,0.1,1e100),
				 ShiftX("ShiftX",-0.5,-1,1),
				 ShiftY("ShiftY",0,-1,1),
				 IterationColorCount("MaxIteration",256,16,2048);

void Fractal(const Posize &ps)
{
	Uint32 *Buffer=new Uint32[ps.w*ps.h];
	
	Uint32 *DeviceData=nullptr;
	hipMalloc((void**)&DeviceData,ps.Size()*sizeof(Uint32));
	for (int i=0;i<(ps.h+1023)/1024;++i)
	{
		dim3 grid(ps.w);
		dim3 threads(min(ps.h-i*1024,1024));
		GetFractalInKernel << <grid, threads >> >(DeviceData,ps.w,ps.h,i,EscapeRadius,Scale,ShiftX,ShiftY,IterationColorCount);
		hipDeviceSynchronize();
	}
	hipMemcpy(Buffer,DeviceData,ps.Size()*sizeof(Uint32),hipMemcpyDeviceToHost);
    hipFree(DeviceData);
	
	SDL_Surface *sur=SDL_CreateRGBSurfaceWithFormatFrom((void*)Buffer,ps.w,ps.h,32,ps.w*4,SDL_PIXELFORMAT_RGBA32);
	SDL_SetSurfaceBlendMode(sur,SDL_BLENDMODE_BLEND);
	CurrentWindow->RenderCopy(CreateTextureFromSurfaceAndDelete(sur),ps.ToOrigin(),ps);
	DELETEtoNULL(Buffer);
}

void FractalInit()
{
	SetFractalName(PUIT("Mandelbrot EscapeTimeAlgorithm"));
	new PosEventLayerI(0,PUI_FA_MAINWINDOW,new PosizeEX_Fa6_Full,
		[](int&,const PUI_PosEvent *event,int mode)->int
		{
			if (event->type==PUI_Event::Event_MouseEvent)
				if (event->MouseEvent()->state&PUI_MouseEvent::Mouse_Left)
				{
					Posize WH=GetFractalDisplayPosize();
					ShiftX.SetX(ShiftX-event->delta.x*2.0/WH.w/Scale),ShiftY.SetX(ShiftY-event->delta.y*2.0/WH.h/Scale);
					return 1;
				}
			return 0;
		},0);
	new EventLayerI(0,PUI_FA_MAINWINDOW,
		[](int&,const PUI_Event *event)->int
		{
			if (event->type==PUI_Event::Event_WheelEvent)
				if (event->WheelEvent()->dy!=0)
					return Scale.SetX(Scale*(event->WheelEvent()->dy<0?0.8:1.2)),1;
			return 0;
		},0);
	EscapeRadius.InitWidgets();
	Scale.InitWidgets();
	ShiftX.InitWidgets();
	ShiftY.InitWidgets();
	IterationColorCount.InitWidgets();
}
